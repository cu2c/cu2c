#define PI 3.141592653589792346
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


// Global CUDA function
void runDev(double *dArr){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    dArr[idx] = (double)idx+PI;
}

//Lorem ipsum dolor sit amet
void main(){
    double *hArr, *dArr;
    size_t dimD = 5 * sizeof(double);
    hipMalloc((void **) &dArr,  dimD);
    hipMemcpy(dArr, hArr, dimD, hipMemcpyHostToDevice);
    <<20,30>> runDev(dArr);
    hipMemcpy(hArr, dArr, dimD, hipMemcpyDeviceToHost);
    free(dArr);
}
