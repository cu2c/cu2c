#include "hip/hip_runtime.h"
#define PI 3.141592653589792346
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void runDev(double *dArr){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    dArr[idx] = (double)idx+PI;
}

//Lorem ipsum dolor sit amet
void main(){
    double *hArr, *dArr, **testD, ********snake;
    int **testI;
    size_t dimD = 5 * sizeof(double);
    size_t dimE, dimC=sizeof(int)*8;
    hipMalloc((void **) &dArr,  dimD);
    hipMemcpy(dArr, hArr, dimD, hipMemcpyHostToDevice); // HOST2DEV
    <<20,30>> runDev(dArr);
    hipMemcpy(hArr, dArr, dimD, hipMemcpyDeviceToHost);
    free(dArr);
}
