#include "hip/hip_runtime.h"
#define PI 3.141592653589792346
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void runDev(double *dArr){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    dArr[idx] = (double)idx+PI;
}

//Lorem ipsum dolor sit amet
void main(){
    double *hArr, *dArr;
    size_t dimD = 5 * sizeof(double);
    hipMalloc((void **) &dArr,  dimD);
    hipMemcpy(dArr, hArr, dimD, hipMemcpyHostToDevice);
/*cuda function is here*/    runDev <<<20, 30 >>> (dArr); /*HERE!*/
    hipMemcpy(hArr, dArr, dimD, hipMemcpyDeviceToHost);
    free(dArr);
}
